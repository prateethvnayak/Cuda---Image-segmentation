#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "sussman_kernel.h"
#include ""
#include <math.h>
#include <stdio.h>
//EPS 0.0001
#define BLOCK_SIZE 16
__global__ void sussman_kernel(float* phidd, float* phid, unsigned int height, unsigned int width);
__global__ void mean_kernel(float* phid, unsigned int* datain_d, unsigned int height, unsigned int width, unsigned int* sum_neg_d, unsigned int* sum_pos_d, int* c_neg_d, int* c_pos_d);
__global__ void force_kernel(float* phid, unsigned int* datain_d, float* F_d, unsigned int height, unsigned int width, double* maxF_d, int* stop_d, unsigned int* mean_neg_d, unsigned int* mean_pos_d, int* c_neg_d, int* c_pos_d);
//void kernel_call(float* phidd, float* phid, unsigned int height, unsigned int width);
//void kernel_call_mean(float* phid, float* datain_d, unsigned int height, unsigned int width, double* mean_neg_d, double* mean_pos_d, int* c_neg_d, int* c_pos_d);
//void kernel_call_force(float* phid, float* datain_d, float* F_d, unsigned int height, unsigned int width, double* maxF_d, int* stop_d, double mean_neg, double mean_pos);

__global__ void set(unsigned int* mean_neg_d, unsigned int* mean_pos_d, int* c_neg_d, int* c_pos_d, double* maxF_d, double* max_dphidt_d)
{
	int tid = threadIdx.x;
	if (tid == 0)
	{
		*mean_neg_d = 0;
		*mean_pos_d = 0;
		*c_neg_d = 0;
		*c_pos_d = 0;
		*maxF_d = 0;
		*max_dphidt_d = 0;
	}
}


__global__ void sussman_kernel_shared(float* phidd, float* phid, unsigned int height, unsigned int width)
{

	float d_phid;
	float a, b, c, d;

	int pixly = threadIdx.x + blockIdx.x * (BLOCK_SIZE-2);
	int pixlx = threadIdx.y + blockIdx.y * (BLOCK_SIZE-2);
	int iny = threadIdx.x + blockIdx.x * (BLOCK_SIZE-2) -1;
	int inx = threadIdx.y + blockIdx.y * (BLOCK_SIZE-2) -1;

        __shared__ float phi_s[BLOCK_SIZE][BLOCK_SIZE];
	
	inx = inx > 0 ? inx : width - 1;
	inx = inx < width ? inx : 0;
	iny = iny > 0 ? iny : height - 1;
	iny = iny < height ? iny : 0;		
	phi_s[threadIdx.x][threadIdx.y] = phid[inx*height + iny];	
	__syncthreads();

	if (pixlx < width && pixly < height)
	{

		int l_x = threadIdx.x;
		int r_x = threadIdx.x+2;
		int m_x = threadIdx.x+1;
		int u_y = threadIdx.y;
		int d_y = threadIdx.y+2;
		int m_y = threadIdx.y+1;
		

		if(threadIdx.x < BLOCK_SIZE-2 && threadIdx.y < BLOCK_SIZE-2 )
		{

			float sussman_sign = phi_s[m_x][m_y]/ sqrtf(phi_s[m_x][m_y]* phi_s[m_x][m_y]+ 1);
			if (phi_s[m_x][m_y]> 0)
			{
				a = fmax((float)(phi_s[m_x][m_y] - phi_s[l_x][m_y]), (float)0);
				b = fmin((float)(phi_s[r_x][m_y] - phi_s[m_x][m_y]), (float)0);
				c = fmax((float)(phi_s[m_x][m_y]- phi_s[m_x][d_y]), (float)0);
				d = fmin((float)(phi_s[m_x][u_y] - phi_s[m_x][m_y]), (float)0);

				d_phid = sqrtf(fmax(a*a, b*b) + fmax(c*c, d*d)) - 1;

			}
			else if (phi_s[m_x][m_y]< 0)
			{
				a = fmin((float)(phi_s[m_x][m_y]- phi_s[l_x][m_y]), (float)0);
				b = fmax((float)(phi_s[r_x][m_y] - phi_s[m_x][m_y]), (float)0);
				c = fmin((float)(phi_s[m_x][m_y]- phi_s[m_x][d_y]), (float)0);
				d = fmax((float)(phi_s[m_x][u_y] - phi_s[m_x][m_y]), (float)0);

				d_phid = sqrtf(fmax(a*a, b*b) + fmax(c*c, d*d)) - 1;

			}
			else
			{
				d_phid = 0;
			}
/* divergence elimination, worse performance
 
			int sign = 2*(phi_s[m_x][m_y]> 0)-1;
			float sussman_sign = phi_s[m_x][m_y]/ sqrtf(phi_s[m_x][m_y]* phi_s[m_x][m_y]+ 1);


				a = sign*fmax((float)sign* (phi_s[m_x][m_y] - phi_s[l_x][m_y]), (float)0);
				b = sign*fmin((float)(sign*phi_s[r_x][m_y] - phi_s[m_x][m_y]), (float)0);
				c = sign*fmax((float)(sign*phi_s[m_x][m_y]- phi_s[m_x][d_y]), (float)0);
				d = sign*fmin((float)(sign*phi_s[m_x][u_y] - phi_s[m_x][m_y]), (float)0);

				d_phid = sqrtf(fmax(a*a, b*b) + fmax(c*c, d*d)) - 1;

			
*/


			phidd[pixlx*height + pixly] = phi_s[threadIdx.x+1][threadIdx.y+1]- 0.5 * sussman_sign * d_phid;
		}
	}
}

__global__ void sussman_kernel(float* phidd, float* phid, unsigned int height, unsigned int width)
{

	
	int r_x, l_x, u_y, d_y;
	float d_phid;
	float a, b, c, d;

	int pixly = threadIdx.x + blockIdx.x * blockDim.x;
	int pixlx = threadIdx.y + blockIdx.y * blockDim.y;
	float plxly = phid[pixlx*height + pixly];
	if (pixlx < width && pixly < height)
	{
		// neighbor indices
		l_x = pixlx - 1 > 0 ? pixlx - 1 : width - 1;
		r_x = pixlx + 1 < width ? pixlx + 1 : 0;
		u_y = pixly - 1 > 0 ? pixly - 1 : height - 1;
		d_y = pixly + 1 < height ? pixly + 1 : 0;

		// sign of the value
		float sussman_sign = plxly / sqrtf(plxly * plxly + 1);

		// calculate difference between neghbor pixels (with thresholding)
		if (plxly > 0)
		{
			a = fmax((float)(plxly - phid[l_x*height + pixly]), (float)0);
			b = fmin((float)(phid[r_x*height + pixly] - plxly), (float)0);
			c = fmax((float)(plxly - phid[pixlx*height + d_y]), (float)0);
			d = fmin((float)(phid[pixlx*height + u_y] - plxly), (float)0);

			d_phid = sqrtf(fmax(a*a, b*b) + fmax(c*c, d*d)) - 1;

		}
		else if (plxly < 0)
		{
			a = fmin((float)(plxly - phid[l_x*height + pixly]), (float)0);
			b = fmax((float)(phid[r_x*height + pixly] - plxly), (float)0);
			c = fmin((float)(plxly - phid[pixlx*height + d_y]), (float)0);
			d = fmax((float)(phid[pixlx*height + u_y] - plxly), (float)0);

			d_phid = sqrtf(fmax(a*a, b*b) + fmax(c*c, d*d)) - 1;

		}
		else
		{
			d_phid = 0;
		}
		// update phi
		phidd[pixlx*height + pixly] = plxly - 0.5 * sussman_sign * d_phid;
	}
}

void sussman_kernel_call(float* phidd, float* phid, unsigned int height, unsigned int width)
{
/*
       //shared memory version
	dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 gridsize((height + BLOCK_SIZE-1) / (BLOCK_SIZE-2), (width + BLOCK_SIZE-1) / (BLOCK_SIZE-2), 1);
	// kernel call
	sussman_kernel_shared << <gridsize, blocksize >> >(phidd, phid, height, width);
*/
        //global memory version
	dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 gridsize((height + BLOCK_SIZE-1) / BLOCK_SIZE, (width + BLOCK_SIZE-1) / BLOCK_SIZE, 1);
	// kernel call
	sussman_kernel << <gridsize, blocksize >> >(phidd, phid, height, width);


}

__device__ double atomicAddDouble(double* address,
	double val) {
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old =
		*address_as_ull, assumed;
	do {
		assumed = old;	// READ
		old = atomicCAS(address_as_ull, assumed,
			val + assumed);	// MODIFY + WRITE
	} while (assumed != old);
	return old;
}


__global__ void mean_kernel(float* phid, unsigned int* datain_d, unsigned int height, unsigned int width, unsigned int* sum_neg_d, unsigned int* sum_pos_d, int* c_neg_d, int* c_pos_d)
{
	__shared__ unsigned int s_sum_neg[32];
	__shared__ unsigned int s_sum_pos[32];
	__shared__ int s_c_neg[32];
	__shared__ int s_c_pos[32];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int tx = threadIdx.x;
	// private copies of sum and counters
	if (tx < 32)
	{
		s_sum_neg[tx] = 0;
		s_sum_pos[tx] = 0;
		s_c_neg[tx] = 0;
		s_c_pos[tx] = 0;
	}
	__syncthreads();
	// calculate sum and count
	if (tid < height*width)
	{
		if (phid[tid] <= 0)
		{
			atomicAdd(&s_sum_neg[tx%32], datain_d[tid]);
			atomicAdd(&s_c_neg[tx%32], 1);
		}
		else
		{
			atomicAdd(&s_sum_pos[tx%32], datain_d[tid]);
			atomicAdd(&s_c_pos[tx%32], 1);
		}

		__syncthreads();
		
		if(tx<32)
		{
			atomicAdd(&s_sum_neg[0], s_sum_neg[tx]);
			atomicAdd(&s_sum_pos[0], s_sum_pos[tx]);
			atomicAdd(&s_c_neg[0], s_c_neg[tx]);
			atomicAdd(&s_c_pos[0], s_c_pos[tx]);


		}

		if (tx == 0)
		{
			// accumulate global sum and counter
			atomicAdd(sum_neg_d, s_sum_neg[0]);
			atomicAdd(c_neg_d, s_c_neg[0]);
			atomicAdd(sum_pos_d, s_sum_pos[0]);
			atomicAdd(c_pos_d, s_c_pos[0]);
		}
	}
}

//For calculation of mean
void kernel_call_mean(float* phid, unsigned int* datain_d, unsigned int height, unsigned int width, unsigned int* mean_neg_d, unsigned int* mean_pos_d, int* c_neg_d, int* c_pos_d, double* maxF_d, double* max_dphidt_d)
{
	dim3 blocksize(256, 1, 1);
	dim3 gridsize((height*width + 255) / 256, 1, 1);
	// kernel call to set
	set << <1, 1 >> > (mean_neg_d, mean_pos_d, c_neg_d, c_pos_d, maxF_d, max_dphidt_d);

	//kernel for mean
	mean_kernel << <gridsize, blocksize >> >(phid, datain_d, height, width, mean_neg_d, mean_pos_d, c_neg_d, c_pos_d);
	// copy back to host
}

__device__ static double atomicMaxDouble(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
	} while (assumed != old);
	return __longlong_as_double(old);
}

__global__ void force_kernel(float* phid, unsigned int* datain_d, float* F_d, unsigned int height, unsigned int width, double* maxF_d, int* stop_d, unsigned int* mean_neg_d, unsigned int* mean_pos_d, int* c_neg_d, int* c_pos_d)
{
	__shared__ double s_maxF[32];
	__shared__ float mean_neg;
	__shared__ float mean_pos;
	//__shared__ int s_stop;

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int tx = threadIdx.x;
	// mean of inside and outside of 0 level set
	if (tx == 0)
	{
		mean_neg = *mean_neg_d / ((float)(*c_neg_d) + 0.00001);
		mean_pos = *mean_pos_d / ((float)(*c_pos_d) + 0.00001);
		
	}
	if (tx < 32)
	{
		s_maxF[tx] = 0;
	}
	__syncthreads();

	float phiVal = phid[tid];
	float dataVal = datain_d[tid];
	if (tid < height*width)
	{
		// calculate the force pushing the boundary
		float val = 0.0f;
		if (phiVal < 1.2 && phiVal > -1.2)
		{
			val = (dataVal - mean_neg) * (dataVal - mean_neg) - (dataVal - mean_pos) * (dataVal - mean_pos);
			atomicMaxDouble(&s_maxF[tx%32], fabs(val));
			F_d[tid] = val;

		}

		__syncthreads();
		if (tx < 32)
		{
			atomicMaxDouble(&s_maxF[0], s_maxF[tx]);
		}
	//	__syncthreads();
		if (tx == 0)
		{
			atomicMaxDouble(maxF_d, fabs(s_maxF[0]));
		}
	}
}
void kernel_call_force(float* phid, unsigned int* datain_d, float* F_d, unsigned int height, unsigned int width, double* maxF_d, int* stop_d, unsigned int* mean_neg_d, unsigned int* mean_pos_d, int* c_neg_d, int* c_pos_d)
{

	dim3 blocksize(256, 1, 1);
	dim3 gridsize((height*width + 255) / 256, 1, 1);
	// kernel call
	force_kernel << <gridsize, blocksize >> >(phid, datain_d, F_d, height, width, maxF_d, stop_d, mean_neg_d, mean_pos_d, c_neg_d, c_pos_d);
	// copy back to host
	//                hipDeviceSynchronize();
}

__global__ void gradient_kernel(float* phid, float* curvature_d, float* F_d, float* dphidt_d, double* max_dphidt_d, double alpha, double* maxF_d, unsigned int height, unsigned int width)
{
	__shared__ double max_dphidt_s;

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int tx = threadIdx.x;

	if (tx == 0)
		max_dphidt_s = 0;

	__syncthreads();


	if (tid < height*width)
	{
		// calculate gradient only near boundary of 0 level set
		if (phid[tid] < 1.2 && phid[tid] > -1.2)
		{
			dphidt_d[tid] = (F_d[tid] / *maxF_d) + alpha * curvature_d[tid];
			atomicMaxDouble(&max_dphidt_s, (double)dphidt_d[tid]);
		}

		__syncthreads();

		// max of gradient for normalization
		if (tx == 0)
		{
			atomicMaxDouble(max_dphidt_d, max_dphidt_s);
		}
	}

}

// Function definition to launch the kernel to compute the gradient
void gradient_kernel_call(float* phid, float* curvature_d, float* F_d, float* dphidt_d, unsigned int height, unsigned int width, double* max_dphidt_d, double alpha, double* maxF_d)
{
	dim3 blocksize(256, 1, 1);
	dim3 gridsize((height*width + 255) / 256, 1, 1);
	gradient_kernel << <gridsize, blocksize >> >(phid, curvature_d, F_d, dphidt_d, max_dphidt_d, alpha, maxF_d, height, width);
}

__global__ void CFL_kernel(float* phidCFL, float* phid, float* dphidt_d, double* max_dphidt_d, unsigned int height, unsigned int width)
{
	__shared__ float dt;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	// calculate step size
	if (threadIdx.x == 0)
	{
		dt = 0.45 / (*max_dphidt_d + 0.00001);
	}
	
	__syncthreads();
	if (tid < height*width)
	{
		// update only pixels near boundary of 0 level set
		if (phid[tid] < 1.2 && phid[tid] > -1.2)
		{
			phidCFL[tid] = phid[tid] + dt * dphidt_d[tid];
		}
		else
		{
			phidCFL[tid] = phid[tid];
		}

	}
}

// Function definition to launch the kernel for CFL condition
void CFL_kernel_call(float* phidCFL, float* phid, float* dphidt_d, unsigned int height, unsigned int width, double *max_dphidt_d)
{
	dim3 blocksize(256, 1, 1);
	dim3 gridsize((height*width + 255) / 256, 1, 1);
	CFL_kernel << <gridsize, blocksize >> >(phidCFL, phid, dphidt_d, max_dphidt_d, height, width);
}

__global__ void curvature_kernel(float* curvature_d, float* phid, unsigned int height, unsigned int width)
{
	float phi_x, phi_y, phi_xx, phi_yy, phi_xy;
	int xm1, ym1, xp1, yp1;

	int tid_y = threadIdx.x + blockIdx.x * blockDim.x;
	int tid_x = threadIdx.y + blockIdx.y * blockDim.y;

	if (tid_x<width&&tid_y<height)
	{
		if (phid[tid_x*height + tid_y] < 1.2 && phid[tid_x*height + tid_y] > -1.2)
		{
			// boundary check
			xm1 = tid_x - 1 < 0 ? 0 : tid_x - 1;
			ym1 = tid_y - 1 < 0 ? 0 : tid_y - 1;
			xp1 = tid_x + 1 >= width ? width - 1 : tid_x + 1;
			yp1 = tid_y + 1 >= height ? height - 1 : tid_y + 1;
			

			// differences between neighbors
			phi_x = -phid[xm1*height + tid_y] + phid[xp1*height + tid_y];
			phi_y = -phid[tid_x*height + ym1] + phid[tid_x*height + yp1];
			phi_xx = phid[xm1*height + tid_y] + phid[xp1*height + tid_y] - 2 * phid[tid_x*height + tid_y];
			phi_yy = phid[tid_x*height + ym1] + phid[tid_x*height + yp1] - 2 * phid[tid_x*height + tid_y];
			phi_xy = 0.25*(-phid[xm1*height + ym1] - phid[xp1*height + yp1] + phid[xp1*height + ym1] + phid[xm1*height + yp1]);
			// curvature calculation
			curvature_d[tid_x*height + tid_y] = phi_x*phi_x * phi_yy + phi_y*phi_y * phi_xx - 2 * phi_x * phi_y * phi_xy;
			curvature_d[tid_x*height + tid_y] = curvature_d[tid_x*height + tid_y] / (phi_x*phi_x + phi_y*phi_y + 0.00001);
		}
		else
		{
			curvature_d[tid_x*height + tid_y] = 0;
		}
	}
}


__global__ void curvature_kernel_shared(float* curvature_d, float* phid, unsigned int height, unsigned int width)
{
	float phi_x, phi_y, phi_xx, phi_yy, phi_xy;
	//int xm1, ym1, xp1, yp1;

        __shared__ float phi_s[BLOCK_SIZE][BLOCK_SIZE];

	
	int tid_y = threadIdx.x + blockIdx.x * (BLOCK_SIZE-2);
	int tid_x = threadIdx.y + blockIdx.y * (BLOCK_SIZE-2);    
	int iny = threadIdx.x + blockIdx.x * (BLOCK_SIZE-2) -1;
	int inx = threadIdx.x + blockIdx.x * (BLOCK_SIZE-2) -1;

	int l_x, r_x, m_x, u_y, d_y, m_y;
	
	inx = inx > 0 ? inx : 0;
	inx = inx < width ? inx : width - 1;
	iny = iny > 0 ? iny : 0;
	iny = iny < height ? iny : height - 1;
		
	phi_s[threadIdx.x][threadIdx.y] = phid[inx*height + iny];	
	__syncthreads();


	if (tid_x<width&&tid_y<height)
	{
		l_x = threadIdx.x;
		r_x = threadIdx.x+2;
		m_x = threadIdx.x+1;
		u_y = threadIdx.y;
		d_y = threadIdx.y+2;
		m_y = threadIdx.y+1;
		if (phi_s[m_x][m_y] < 1.2 && phi_s[m_x][m_y] > -1.2)
		{
			// boundary check


			// differences between neighbors
			phi_x = -phi_s[l_x][m_y] + phi_s[r_x][m_y];
			phi_y = -phi_s[m_x][u_y] + phi_s[m_x][d_y];
			phi_xx = phi_s[l_x][m_y] + phi_s[r_x][m_y] - 2 * phi_s[m_x][m_y];
			phi_yy = phi_s[m_x][u_y] + phi_s[m_x][d_y] - 2 * phi_s[m_x][m_y];
			phi_xy = 0.25*(-phi_s[l_x][u_y] - phi_s[r_x][d_y] + phi_s[r_x][u_y] + phi_s[l_x][d_y]);
			// curvature calculation
			curvature_d[tid_x*height + tid_y] = phi_x*phi_x * phi_yy + phi_y*phi_y * phi_xx - 2 * phi_x * phi_y * phi_xy;
			curvature_d[tid_x*height + tid_y] = curvature_d[tid_x*height + tid_y] / (phi_x*phi_x + phi_y*phi_y + 0.00001);
		}
		else
		{
			curvature_d[tid_x*height + tid_y] = 0;
		}
	}
}


void curvature_kernel_call(float* curvature_d, float* phid, unsigned int height, unsigned int width)
{
/*
	// global memory version
	dim3 blocksize(16, 16, 1);
	dim3 gridsize((height + 15) / 16, (width + 15) / 16, 1);
	// kernel call
	curvature_kernel << <gridsize, blocksize >> >(curvature_d, phid, height, width);
*/	
	// shared memory version
	dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 gridsize((height + BLOCK_SIZE-1) / (BLOCK_SIZE-2), (width + BLOCK_SIZE-1) / (BLOCK_SIZE-2), 1);

	// kernel call
	curvature_kernel_shared << <gridsize, blocksize >> >(curvature_d, phid, height, width);

	

}